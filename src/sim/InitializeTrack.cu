#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file InitializeTrack.cu
//---------------------------------------------------------------------------//
#include "InitializeTrack.cuh"

namespace celeritas
{
//---------------------------------------------------------------------------//
__global__ initialize_tracks(span<const Primary>          primaries,
                             const ParticleParamsPointers pparams,
                             const ParticleStatePointers  pstates,
                             const GeoParamsPointers      gparams,
                             const GeoStatePointers       gstates,
                             ThreadId starting_thread_offset)
{
    auto local_thread_id = celeritas::KernelParamCalculator::thread_id();
    if (!(local_thread_id < primaries.size()))
        return;
    const Primary& primary = primaries[local_thread_id.get()];
    ThreadId       offset_thread_id{starting_thread_offset.get()
                              + local_thread_id.get()};

    // TODO: Initialize sim state
    {}

    // Initialize particle physics data
    {
        ParticleTrackView particle(pparams, pstates, offset_thread_id);
        particle = primary.particle;
    }

    // Initialize geometry state
    {
        GeoTrackView geo(gparams, gstates, offset_thread_id);
        geo = primary.geo;
    }
}

//---------------------------------------------------------------------------//
} // namespace celeritas
