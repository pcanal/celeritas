#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file InitializeTrack.cu
//---------------------------------------------------------------------------//
#include "InitializeTrack.cuh"

#include <algorithm>
#include <vector>
#include <thrust/device_vector.h>
#include "base/KernelParamCalculator.cuda.hh"

namespace celeritas
{
//---------------------------------------------------------------------------//
__global__ initialize_tracks(size_type                    size,
                             TrackInitializer*            initializers,
                             size_type*                   vacancies,
                             //const SimParamsPointers      sparams,
                             //const SimStatePointers       sstates,
                             const ParticleParamsPointers pparams,
                             const ParticleStatePointers  pstates,
                             const GeoParamsPointers      gparams,
                             const GeoStatePointers       gstates)
{
    auto thread_id = KernelParamCalculator::thread_id();
    if (thread_id < size)
    {
        // Index of the empty slot in the track states
        ThreadId slot_id = ThreadId(vacancies[thread_id.get()]);

        // Index of the initializer used to fill the empty slot. thread id 0
        // will use the last track initializer in the vector, thread id 1 the
        // next to last, etc.
        size_type init_id = size - thread_id.get() + 1;

        const TrackInitializer& initializer = initializers[init_id];
 
        // Initialize sim state
        {
            //SimTrackView sim(sparams, sstates, slot_id);
            //sim = initializer.sim;
        }
 
        // Initialize particle physics data
        {
            ParticleTrackView particle(pparams, pstates, slot_id);
            particle = initializer.particle;
        }
 
        // Initialize geometry state
        {
            GeoTrackView geo(gparams, gstates, slot_id);
            geo = initializer.geo;
        }
    }
}

//---------------------------------------------------------------------------//
/*!
 *
 */
void pop_tracks(thrust::device_vector<const TrackInitializer>& initializers,
                thrust::device_vector<size_type>&              vacancies,
                //const SimParamsPointers                        sparams,
                //const SimStatePointers                         sstates,
                const ParticleParamsPointers                   pparams,
                const ParticleStatePointers                    pstates,
                const GeoParamsPointers                        gparams,
                const GeoStatePointers                         gstates)
{
    // Number of tracks to initialize
    size_type num_new_tracks = std::min(initializers.size(), vacancies.size());

    // If there are more tracks than vacant slots, resize the vacancies to the
    // number of tracks available
    vacancies.resize(num_new_tracks);

    // Launch kernel to create new tracks from track initializers
    KernelParamCalculator calc_launch_params;
    auto params = calc_launch_params(num_new_tracks);
    initialize_tracks<<<params.grid_size, params.block_size>>>(
        num_new_tracks, thrust::raw_pointer_cast(initializers.data()),
        thrust::raw_pointer_cast(vacancies.data()), //sparams, sstates,
        pparams, pstates, gparams, gstates);

    // Resize the vector of track initializers to hold the tracks that have not
    // been created yet
    size_type size = initializers.size() - num_new_tracks;
    initializers.resize(size);
}

//---------------------------------------------------------------------------//
} // namespace celeritas
