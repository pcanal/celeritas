#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file InitializeTracks.cu
//---------------------------------------------------------------------------//
#include "InitializeTracks.hh"

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/remove.h>
#include <thrust/scan.h>
#include <vector>
#include "base/Atomics.hh"
#include "base/DeviceVector.hh"

namespace celeritas
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Initialize the track states on device. The track initializers are created
 * from either primary particles or secondaries. The new tracks are inserted
 * into empty slots (vacancies) in the track vector.
 */
__global__ void
initialize_tracks_kernel(const StatePointers            states,
                         const ParamPointers            params,
                         const TrackInitializerPointers initializers)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < initializers.vacancies.size())
    {
        // Get the track initializer, starting from the back of the vector
        const TrackInitializer& init
            = initializers.tracks[initializers.tracks.size() - thread_id - 1];

        // Index of the empty slot to create the new track in
        size_type empty_id = initializers.vacancies[thread_id];

        // Initialize particle physics data
        ParticleTrackView particle(
            params.particle, states.particle, ThreadId(empty_id));
        particle = init.particle;

        // Initialize geometry state
        // GeoTrackView geo(params.geo, states.geo, ThreadId(empty_id));
        // geo = init.geo;

        // Initialize simulation state
        SimTrackView sim(states.sim, ThreadId(empty_id));
        sim = init.sim;
    }
}

//---------------------------------------------------------------------------//
/*!
 * Find empty slots in the track vector and count the number of secondaries
 * that survived cutoffs for each interaction.
 */
__global__ void
find_vacancies_kernel(const StatePointers            states,
                      const TrackInitializerPointers initializers)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < states.size())
    {
        // Determine which indices in the track states are available for
        // initializing new particles
        SimTrackView sim(states.sim, ThreadId(thread_id));
        if (sim.alive())
        {
            initializers.vacancies[thread_id] = occupied_flag();
        }
        else
        {
            initializers.vacancies[thread_id] = thread_id;
        }

        // Count how many secondaries survived cutoffs for each track
        initializers.secondary_counts[thread_id] = 0;
        const Interaction& result = states.interactions[thread_id];
        for (const auto& secondary : result.secondaries)
        {
            if (secondary)
            {
                ++initializers.secondary_counts[thread_id];
            }
        }
    }
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers on device from primary particles.
 */
__global__ void
process_primaries_kernel(const span<const Primary>      primaries,
                         const TrackInitializerPointers initializers)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < primaries.size())
    {
        size_type         offset_id = initializers.tracks.size();
        TrackInitializer& init = initializers.tracks[offset_id + thread_id];

        // Construct a track initializer from a primary particle
        init = primaries[thread_id];
    }
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers on device from secondary particles.
 */
__global__ void
process_secondaries_kernel(const StatePointers            states,
                           const ParamPointers            params,
                           const TrackInitializerPointers initializers)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < states.size())
    {
        // Construct the state accessors
        // GeoTrackView geo(params.geo, states.geo, ThreadId(thread_id));
        SimTrackView sim(states.sim, ThreadId(thread_id));

        // Starting index in the vector of track initializers
        size_type index = initializers.secondary_counts[thread_id];

        const Interaction& result = states.interactions[thread_id];
        for (const auto& secondary : result.secondaries)
        {
            // If the secondary survived cutoffs
            if (secondary)
            {
                // Construct a track initializer from a secondary
                TrackInitializer& init = initializers.tracks[index];
                init.particle.def_id   = secondary.def_id;
                init.particle.energy   = secondary.energy;
                init.geo.dir           = secondary.direction;
                // init.geo.pos           = geo.pos();
                unsigned int track_id = initializers.track_count + 1 + index++;
                init.sim.track_id     = TrackId{track_id};
                init.sim.parent_id    = sim.track_id();
                init.sim.event_id     = sim.event_id();
                init.sim.alive        = true;
            }
        }
    }
}

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
/*!
 * Initialize the track states on device.
 */
void initialize_tracks(StatePointers          states,
                       ParamPointers          params,
                       TrackInitializerStore& initializers)
{
    // The number of new tracks to initialize is the smaller of the number of
    // empty slots in the track vector and the number of track initializers
    size_type num_new_tracks
        = std::min(initializers.num_vacancies(), initializers.size());
    initializers.num_vacancies() = num_new_tracks;

    // Initialize tracks on device
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(num_new_tracks);
    initialize_tracks_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, params, initializers.device_pointers());

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Resize the vector of track initializers
    initializers.resize(initializers.size() - num_new_tracks);
    initializers.track_count() += num_new_tracks;
}

//---------------------------------------------------------------------------//
/*!
 * Find empty slots in the vector of tracks and count the number of secondaries
 * that survived cutoffs for each interaction.
 */
void find_vacancies(StatePointers states, TrackInitializerStore& initializers)
{
    // Resize the vector of vacancies to be equal to the number of tracks
    initializers.num_vacancies() = states.size();

    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(states.size());
    find_vacancies_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, initializers.device_pointers());

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Remove all the elements in the vacancy vector that were flagged as
    // active tracks, so we are left with a vector containing the (sorted)
    // indices of the empty slots
    span<size_type> vacancies = initializers.device_pointers().vacancies;
    thrust::device_ptr<size_type> end = thrust::remove_if(
        thrust::device_pointer_cast(vacancies.data()),
        thrust::device_pointer_cast(vacancies.data() + vacancies.size()),
        occupied(occupied_flag()));

    // Resize the vector of vacancies to be equal to the number of empty slots
    initializers.num_vacancies() = thrust::raw_pointer_cast(end)
                                   - vacancies.data();
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers from primary particles.
 */
void process_primaries(span<const Primary>    primaries,
                       TrackInitializerStore& initializers)
{
    REQUIRE(primaries.size() <= initializers.capacity() - initializers.size());

    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(primaries.size());
    process_primaries_kernel<<<lparams.grid_size, lparams.block_size>>>(
        primaries, initializers.device_pointers());

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Resize the vector of track initializers
    initializers.resize(initializers.size() + primaries.size());
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers from secondary particles.
 */
void process_secondaries(StatePointers          states,
                         ParamPointers          params,
                         TrackInitializerStore& initializers)
{
    // Sum the total number secondaries produced in all interactions
    span<size_type> counts = initializers.device_pointers().secondary_counts;
    size_type       num_secondaries = thrust::reduce(
        thrust::device_pointer_cast(counts.data()),
        thrust::device_pointer_cast(counts.data()) + counts.size(),
        0,
        thrust::plus<size_type>());

    // TODO: if we don't have space for all the secondaries, we will need to
    // buffer the current track initializers
    REQUIRE(num_secondaries <= initializers.capacity() - initializers.size());

    // The exclusive prefix sum of the number of secondaries produced in each
    // interaction is used to get the starting index in the vector of track
    // initializers for creating initializers from secondaries from an
    // interaction
    thrust::exclusive_scan(
        thrust::device_pointer_cast(counts.data()),
        thrust::device_pointer_cast(counts.data()) + counts.size(),
        counts.data(),
        0);

    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(states.size());
    process_secondaries_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, params, initializers.device_pointers());

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Resize the vector of track initializers
    initializers.resize(initializers.size() + num_secondaries);
}

//---------------------------------------------------------------------------//
} // namespace celeritas
