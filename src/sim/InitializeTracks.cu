#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file InitializeTracks.cu
//---------------------------------------------------------------------------//
#include "InitializeTracks.hh"

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/remove.h>
#include <thrust/scan.h>
#include <vector>
#include "base/Atomics.hh"
#include "base/DeviceVector.hh"

namespace celeritas
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Initialize the track states on device. The track initializers are created
 * from either primary particles or secondaries. The new tracks are inserted
 * into empty slots (vacancies) in the track vector.
 */
__global__ void process_tracks_kernel(const StatePointers            states,
                                      const ParamPointers            params,
                                      const TrackInitializerPointers inits)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < inits.vacancies.size())
    {
        // Get the track initializer, starting from the back of the vector
        const TrackInitializer& track
            = inits.initializers[inits.initializers.size() - thread_id - 1];

        // Index of the empty slot to create the new track in
        size_type empty_id = inits.vacancies[thread_id];

        // Initialize particle physics data
        ParticleTrackView particle(
            params.particle, states.particle, ThreadId(empty_id));
        particle = track.particle;

        // Initialize geometry state
        // GeoTrackView geo(params.geo, states.geo, ThreadId(empty_id));
        // geo = track.geo;

        // Initialize simulation state
        SimTrackView sim(states.sim, ThreadId(empty_id));
        sim = track.sim;
    }
}

//---------------------------------------------------------------------------//
/*!
 * Find empty slots in the track vector and count the number of secondaries
 * that survived cutoffs for each interaction.
 */
__global__ void
process_post_interaction_kernel(const StatePointers            states,
                                const TrackInitializerPointers inits)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < states.size())
    {
        // Determine which indices in the track states are available for
        // initializing new particles
        SimTrackView sim(states.sim, ThreadId(thread_id));
        if (sim.alive())
        {
            inits.vacancies[thread_id] = occupied_flag();
        }
        else
        {
            inits.vacancies[thread_id] = thread_id;
        }

        // Count how many secondaries survived cutoffs for each track
        inits.secondary_counts[thread_id] = 0;
        const Interaction& result = states.interactions[thread_id];
        for (const auto& secondary : result.secondaries)
        {
            if (secondary)
            {
                ++inits.secondary_counts[thread_id];
            }
        }
    }
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers on device from primary particles.
 */
__global__ void process_primaries_kernel(const span<const Primary> primaries,
                                         const TrackInitializerPointers inits)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < primaries.size())
    {
        size_type         offset_id = inits.initializers.size();
        TrackInitializer& track = inits.initializers[offset_id + thread_id];

        // Construct a track initializer from a primary particle
        track = primaries[thread_id];
    }
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers on device from secondary particles.
 */
__global__ void process_secondaries_kernel(const StatePointers states,
                                           const ParamPointers params,
                                           const TrackInitializerPointers inits)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < states.size())
    {
        // Construct the state accessors
        // GeoTrackView geo(params.geo, states.geo, ThreadId(thread_id));
        SimTrackView sim(states.sim, ThreadId(thread_id));

        // Starting index in the vector of track initializers
        size_type index = inits.secondary_counts[thread_id];

        const Interaction& result = states.interactions[thread_id];
        for (const auto& secondary : result.secondaries)
        {
            // If the secondary survived cutoffs
            if (secondary)
            {
                // Construct a track initializer from a secondary
                TrackInitializer& track = inits.initializers[index];
                track.particle.def_id   = secondary.def_id;
                track.particle.energy   = secondary.energy;
                track.geo.dir           = secondary.direction;
                // track.geo.pos           = geo.pos();
                unsigned int track_id = inits.track_count + 1 + index++;
                track.sim.track_id    = TrackId{track_id};
                track.sim.parent_id   = sim.track_id();
                track.sim.event_id    = sim.event_id();
                track.sim.alive       = true;
            }
        }
    }
}

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
/*!
 * Initialize the track states on device.
 */
void process_tracks(StatePointers            states,
                    ParamPointers            params,
                    TrackInitializerPointers inits)
{
    // Initialize tracks on device
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(inits.vacancies.size());
    process_tracks_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, params, inits);

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
/*!
 * Find empty slots in the vector of tracks and count the number of secondaries
 * that survived cutoffs for each interaction.
 */
void process_post_interaction(StatePointers            states,
                              TrackInitializerPointers inits)
{
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(states.size());
    process_post_interaction_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, inits);

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers from primary particles.
 */
void process_primaries(span<const Primary>      primaries,
                       TrackInitializerPointers inits)
{
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(primaries.size());
    process_primaries_kernel<<<lparams.grid_size, lparams.block_size>>>(
        primaries, inits);

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers from secondary particles.
 */
void process_secondaries(StatePointers            states,
                         ParamPointers            params,
                         TrackInitializerPointers inits)
{
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(states.size());
    process_secondaries_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, params, inits);

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
/*!
 * Remove all elements in the vacancy vector that were flagged as active
 * tracks.
 */
size_type remove_occupied(span<size_type> vacancies)
{
    thrust::device_ptr<size_type> end = thrust::remove_if(
        thrust::device_pointer_cast(vacancies.data()),
        thrust::device_pointer_cast(vacancies.data() + vacancies.size()),
        occupied(occupied_flag()));

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // New size of the vacancy vector
    size_type result = thrust::raw_pointer_cast(end) - vacancies.data();

    return result;
}

//---------------------------------------------------------------------------//
/*!
 * Sum the total number of surviving secondaries.
 */
size_type reduce_counts(span<size_type> counts)
{
    size_type result = thrust::reduce(
        thrust::device_pointer_cast(counts.data()),
        thrust::device_pointer_cast(counts.data()) + counts.size(),
        size_type(0),
        thrust::plus<size_type>());

    CELER_CUDA_CALL(hipDeviceSynchronize());
    return result;
}

//---------------------------------------------------------------------------//
/*!
 * Calculate the exclusive prefix sum of the number of surviving secondaries
 * from each interaction.
 */
void exclusive_scan_counts(span<size_type> counts)
{
    thrust::exclusive_scan(
        thrust::device_pointer_cast(counts.data()),
        thrust::device_pointer_cast(counts.data()) + counts.size(),
        counts.data(),
        size_type(0));

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
} // namespace celeritas
