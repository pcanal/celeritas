#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file InitializeTracks.cu
//---------------------------------------------------------------------------//
#include "InitializeTracks.hh"

#include "TrackInitializerPointers.hh"
#include "base/Atomics.hh"
#include "thrust/host_vector.h"
#include "thrust/scan.h"
#include "thrust/sort.h"
#include "thrust/fill.h"

namespace celeritas
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Initialize the track states on device. The track initializers are created
 * from either primary particles or secondaries. The new tracks are inserted
 * into empty slots (vacancies) in the track vector.
 */
__global__ void
initialize_tracks_kernel(size_type                      num_tracks,
                         const unsigned long long int*  vacancies,
                         const TrackInitializerPointers initializers,
                         // const SimParamsPointers        sparams,
                         // const SimStatePointers         sstates,
                         const ParticleParamsPointers pparams,
                         const ParticleStatePointers  pstates,
                         const GeoParamsPointers      gparams,
                         const GeoStatePointers       gstates)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < num_tracks)
    {
        // Get the track initializer, starting from the back of the vector
        size_type               init_id = *initializers.size - thread_id - 1;
        const TrackInitializer& init    = initializers.storage[init_id];

        // Index of the empty slot to create the new track in
        size_type slot_id = vacancies[thread_id];

        // Initialize sim state
        // SimTrackView sim(sparams, sstates, ThreadId(slot_id));
        // sim = init.sim;

        // Initialize particle physics data
        ParticleTrackView particle(pparams, pstates, ThreadId(slot_id));
        particle = init.particle;

        // Initialize geometry state
        // GeoTrackView geo(gparams, gstates, ThreadId(slot_id));
        // geo = init.geo;
    }
}

//---------------------------------------------------------------------------//
/*!
 * Find empty slots in the track vector.
 */
__global__ void find_vacancies_kernel(span<const Interaction> interactions,
                                      unsigned long long int* num_vacancies,
                                      unsigned long long int* vacancies)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < interactions.size())
    {
        const Interaction& result = interactions[thread_id];

        // Determine which indices in the track states are available for
        // initializing new particles
        if (action_killed(result.action))
        {
            unsigned long long int index = atomic_add(num_vacancies, 1ull);
            vacancies[index]             = thread_id;
        }
    }
}

//---------------------------------------------------------------------------//
/*!
 * Count the number of secondaries that survived cutoffs for each interaction.
 */
__global__ void count_secondaries_kernel(span<const Interaction> interactions,
                                         size_type* num_secondaries)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < interactions.size())
    {
        const Interaction& result = interactions[thread_id];

        // Count how many secondaries survived cutoffs for each track
        for (auto secondary : result.secondaries)
        {
            if (secondary.energy > 0)
            {
                ++num_secondaries[thread_id];
            }
        }
    }
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers on device from primary particles.
 */
__global__ void
primary_initializers_kernel(span<const Primary>      primaries,
                            TrackInitializerPointers initializers)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < primaries.size())
    {
        size_type         offset_id = *initializers.size;
        TrackInitializer& init = initializers.storage[offset_id + thread_id];

        // Create a new track initializer from a primary particle
        init = primaries[thread_id];
    }
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers on device from secondary particles.
 */
__global__ void
secondary_initializers_kernel(size_type*               cum_secondaries,
                              span<const Interaction>  interactions,
                              TrackInitializerPointers initializers)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < interactions.size())
    {
        const Interaction& result = interactions[thread_id];

        // Starting index in the vector of track initializers
        size_type index = cum_secondaries[thread_id];

        for (auto secondary : result.secondaries)
        {
            // If the secondary survived cutoffs
            if (secondary.energy > 0)
            {
                // TODO: right now only copying energy
                TrackInitializer& init = initializers.storage[index];
                init.particle.energy   = secondary.energy;
                ++index;
            }
        }
    }
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers on device from primary particles.
 */
//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
/*!
 * Initialize the track states on device.
 */
void initialize_tracks(device_vector<unsigned long long int>& vacancies,
                       TrackInitializerStore&                 storage,
                       // const SimParamsPointers                sparams,
                       // const SimStatePointers                 sstates,
                       const ParticleParamsPointers pparams,
                       const ParticleStatePointers  pstates,
                       const GeoParamsPointers      gparams,
                       const GeoStatePointers       gstates)
{
    // Resize vacancy vector to the number of tracks to be inserted
    size_type count = std::min(vacancies.size(), storage.get_size());
    vacancies.resize(count);

    // Initialize tracks on device
    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(count);
    initialize_tracks_kernel<<<params.grid_size, params.block_size>>>(
        count,
        thrust::raw_pointer_cast(vacancies.data()),
        storage.device_pointers(),
        // sparams,
        // sstates,
        pparams,
        pstates,
        gparams,
        gstates);

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Resize the vector of track initializers
    storage.resize(storage.get_size() - count);
}

//---------------------------------------------------------------------------//
/*!
 * Find empty slots in the vector of tracks
 */
void find_vacancies(span<const Interaction>                interactions,
                    device_vector<unsigned long long int>& num_vacancies,
                    device_vector<unsigned long long int>& vacancies)
{
    vacancies.resize(interactions.size());
    num_vacancies.resize(1);
    thrust::fill(num_vacancies.begin(), num_vacancies.end(), 0);

    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(interactions.size());
    find_vacancies_kernel<<<params.grid_size, params.block_size>>>(
        interactions,
        thrust::raw_pointer_cast(num_vacancies.data()),
        thrust::raw_pointer_cast(vacancies.data()));

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Get the number of empty slots via host-device copy and resize the
    // vacancy vector
    thrust::host_vector<unsigned long long int> size = num_vacancies;
    vacancies.resize(size.front());

    // Sort the indices of the empty slots
    thrust::sort(vacancies.begin(), vacancies.end());
}

//---------------------------------------------------------------------------//
/*!
 * Count the number of secondaries that survived cutoffs for each interaction.
 */
void count_secondaries(span<const Interaction>   interactions,
                       device_vector<size_type>& num_secondaries)
{
    // Resize and reset count
    num_secondaries.resize(interactions.size());
    thrust::fill(num_secondaries.begin(), num_secondaries.end(), 0);

    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(interactions.size());
    count_secondaries_kernel<<<params.grid_size, params.block_size>>>(
        interactions, thrust::raw_pointer_cast(num_secondaries.data()));

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers from primary particles
 */
void primary_initializers(span<const Primary>    primaries,
                          TrackInitializerStore& storage)
{
    REQUIRE(primaries.size() <= storage.capacity() - storage.get_size());

    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(primaries.size());
    primary_initializers_kernel<<<params.grid_size, params.block_size>>>(
        primaries, storage.device_pointers());

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Resize the vector of track initializers
    storage.resize(storage.get_size() + primaries.size());
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers from secondary particles
 */
void secondary_initializers(device_vector<size_type>& num_secondaries,
                            span<const Interaction>   interactions,
                            TrackInitializerStore&    storage)
{
    REQUIRE(interactions.size() == num_secondaries.size());

    // The exclusive prefix sum of the number of secondaries produced in each
    // interaction is used to get the starting index in the vector of track
    // initializers for creating initializers from secondaries from an
    // interaction
    size_type count = num_secondaries.back();
    thrust::exclusive_scan(num_secondaries.begin(),
                           num_secondaries.end(),
                           num_secondaries.begin(),
                           0);
    count += num_secondaries.back();

    REQUIRE(count <= storage.capacity() - storage.get_size());

    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(interactions.size());
    secondary_initializers_kernel<<<params.grid_size, params.block_size>>>(
        thrust::raw_pointer_cast(num_secondaries.data()),
        interactions,
        storage.device_pointers());

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Resize the vector of track initializers
    storage.resize(storage.get_size() + count);
}

//---------------------------------------------------------------------------//
} // namespace celeritas
