#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file InitializeTracks.cu
//---------------------------------------------------------------------------//
#include "InitializeTracks.hh"

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/remove.h>
#include <thrust/scan.h>
#include <vector>
#include "base/Atomics.hh"
#include "base/DeviceVector.hh"

namespace celeritas
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
/*!
 * Initialize the track states on device. The track initializers are created
 * from either primary particles or secondaries. The new tracks are inserted
 * into empty slots (vacancies) in the track vector.
 *
 * TODO: Add sim states
 */
__global__ void
initialize_tracks_kernel(const span<const size_type>  vacancies,
                         const span<TrackInitializer> initializers,
                         const ParticleParamsPointers pparams,
                         const ParticleStatePointers  pstates,
                         const GeoParamsPointers      gparams,
                         const GeoStatePointers       gstates)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < vacancies.size())
    {
        // Get the track initializer, starting from the back of the vector
        size_type               init_id = initializers.size() - thread_id - 1;
        const TrackInitializer& init    = initializers[init_id];

        // Index of the empty slot to create the new track in
        size_type slot_id = vacancies[thread_id];

        // Initialize particle physics data
        ParticleTrackView particle(pparams, pstates, ThreadId(slot_id));
        particle = init.particle;

        // Initialize geometry state
        // GeoTrackView geo(gparams, gstates, ThreadId(slot_id));
        // geo = init.geo;
    }
}

//---------------------------------------------------------------------------//
/*!
 * Find empty slots in the track vector.
 */
__global__ void find_vacancies_kernel(span<size_type>         vacancies,
                                      span<const Interaction> interactions)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < interactions.size())
    {
        const Interaction& result = interactions[thread_id];

        // Determine which indices in the track states are available for
        // initializing new particles
        if (action_killed(result.action))
        {
            vacancies[thread_id] = thread_id;
        }
        else
        {
            // Flag as a track that's still alive
            vacancies[thread_id] = occupied_flag();
        }
    }
}

//---------------------------------------------------------------------------//
/*!
 * Count the number of secondaries that survived cutoffs for each interaction.
 */
__global__ void count_secondaries_kernel(size_type* secondary_count,
                                         span<const Interaction> interactions)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < interactions.size())
    {
        secondary_count[thread_id] = 0;
        const Interaction& result = interactions[thread_id];

        // Count how many secondaries survived cutoffs for each track
        for (const auto& secondary : result.secondaries)
        {
            if (secondary.energy.value() > 0)
            {
                ++secondary_count[thread_id];
            }
        }
    }
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers on device from primary particles.
 */
__global__ void
create_from_primaries_kernel(span<const Primary>    primaries,
                             span<TrackInitializer> initializers)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < primaries.size())
    {
        size_type         offset_id = initializers.size();
        TrackInitializer& init      = initializers[offset_id + thread_id];

        // Create a new track initializer from a primary particle
        init = primaries[thread_id];
    }
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers on device from secondary particles.
 */
__global__ void
create_from_secondaries_kernel(size_type*              cum_secondaries,
                               span<const Interaction> interactions,
                               span<TrackInitializer>  initializers)
{
    auto thread_id = KernelParamCalculator::thread_id().get();
    if (thread_id < interactions.size())
    {
        const Interaction& result = interactions[thread_id];

        // Starting index in the vector of track initializers
        size_type index = cum_secondaries[thread_id];

        for (const auto& secondary : result.secondaries)
        {
            // If the secondary survived cutoffs
            if (secondary.energy.value() > 0)
            {
                TrackInitializer& init = initializers[index++];

                // Create a new track initializer from a secondary
                init = secondary;
            }
        }
    }
}

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
/*!
 * Initialize the track states on device.
 */
void initialize_tracks(VacancyStore&          vacancies,
                       TrackInitializerStore& initializers,
                       const ParticleParamsPointers pparams,
                       const ParticleStatePointers  pstates,
                       const GeoParamsPointers      gparams,
                       const GeoStatePointers       gstates)
{
    // The number of new tracks to initialize is the smaller of the number of
    // empty slots in the track vector and the number of track initializers
    size_type num_tracks = std::min(vacancies.size(), initializers.size());
    vacancies.resize(num_tracks);

    // Initialize tracks on device
    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(num_tracks);
    initialize_tracks_kernel<<<params.grid_size, params.block_size>>>(
        vacancies.device_pointers(),
        initializers.device_pointers(),
        pparams,
        pstates,
        gparams,
        gstates);

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Resize the vector of track initializers
    initializers.resize(initializers.size() - num_tracks);
}

//---------------------------------------------------------------------------//
/*!
 * Find empty slots in the vector of tracks
 */
void find_vacancies(VacancyStore&           vacancies,
                    span<const Interaction> interactions)
{
    // Resize the vector of vacancies to be equal to the number of tracks
    size_type num_tracks = interactions.size();
    vacancies.resize(num_tracks);

    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(num_tracks);
    find_vacancies_kernel<<<params.grid_size, params.block_size>>>(
        vacancies.device_pointers(), interactions);

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Remove all the elements in the vacancy vector that were flagged as
    // active tracks, so we are left with a vector containing the (sorted)
    // indices of the empty slots
    thrust::device_ptr<size_type> end = thrust::remove_if(
        thrust::device_pointer_cast(vacancies.device_pointers().data()),
        thrust::device_pointer_cast(vacancies.device_pointers().data()
                                    + vacancies.size()),
        is_not_vacant(occupied_flag()));

    // Resize the vector of vacancies to be equal to the number of empty slots
    vacancies.resize(thrust::raw_pointer_cast(end)
                     - vacancies.device_pointers().data());
}

//---------------------------------------------------------------------------//
/*!
 * Count the number of secondaries that survived cutoffs for each interaction.
 */
void count_secondaries(span<size_type>         secondary_count,
                       span<const Interaction> interactions)
{
    REQUIRE(interactions.size() == secondary_count.size());

    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(interactions.size());
    count_secondaries_kernel<<<params.grid_size, params.block_size>>>(
        secondary_count.data(), interactions);

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers from primary particles
 */
void create_from_primaries(span<const Primary>    primaries,
                           TrackInitializerStore& initializers)
{
    REQUIRE(primaries.size() <= initializers.capacity() - initializers.size());

    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(primaries.size());
    create_from_primaries_kernel<<<params.grid_size, params.block_size>>>(
        primaries, initializers.device_pointers());

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Resize the vector of track initializers
    initializers.resize(initializers.size() + primaries.size());
}

//---------------------------------------------------------------------------//
/*!
 * Create track initializers from secondary particles
 */
void create_from_secondaries(span<size_type>         secondary_count,
                             span<const Interaction> interactions,
                             TrackInitializerStore&  initializers)
{
    REQUIRE(secondary_count.size() == interactions.size());

    // Sum the total number secondaries produced in all interactions
    size_type num_secondaries
        = thrust::reduce(thrust::device_pointer_cast(secondary_count.data()),
                         thrust::device_pointer_cast(secondary_count.data())
                             + secondary_count.size(),
                         0,
                         thrust::plus<size_type>());

    REQUIRE(num_secondaries <= initializers.capacity() - initializers.size());

    // The exclusive prefix sum of the number of secondaries produced in each
    // interaction is used to get the starting index in the vector of track
    // initializers for creating initializers from secondaries from an
    // interaction
    thrust::exclusive_scan(thrust::device_pointer_cast(secondary_count.data()),
                           thrust::device_pointer_cast(secondary_count.data())
                               + secondary_count.size(),
                           secondary_count.data(),
                           0);

    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(interactions.size());
    create_from_secondaries_kernel<<<params.grid_size, params.block_size>>>(
        secondary_count.data(), interactions, initializers.device_pointers());

    CELER_CUDA_CALL(hipDeviceSynchronize());

    // Resize the vector of track initializers
    initializers.resize(initializers.size() + num_secondaries);
}

//---------------------------------------------------------------------------//
} // namespace celeritas
