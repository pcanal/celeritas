#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file TrackInit.test.cu
//---------------------------------------------------------------------------//
#include "InitializeTracks.test.hh"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/binary_search.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include "base/KernelParamCalculator.cuda.hh"
#include "physics/base/SecondaryAllocatorStore.hh"
#include "sim/TrackInitializerPointers.hh"
#include "sim/TrackInitializerStore.hh"
#include "sim/InitializeTracks.hh"
#include "gtest/Main.hh"
#include "gtest/Test.hh"

using namespace celeritas;

namespace celeritas_test
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
__global__ void interact(ITTestInput input, Interaction* interactions)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id();
    if (thread_id.get() < input.num_threads)
    {
        ParticleTrackView particle(input.pparams, input.pstates, thread_id);
        SecondaryAllocatorView allocate(input.sa_view);

        // Allow the particle to interact and create secondaries
        Interactor interact(particle, allocate);
        interactions[thread_id.get()] = interact(thread_id.get() % 3 == 0);
    }
}

__global__ void test_tracks(ITTestInput input, double* output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id();
    if (thread_id.get() < input.num_threads)
    {
        ParticleTrackView particle(input.pparams, input.pstates, thread_id);
        output += thread_id.get();
        *output = particle.energy();
    }
}

__global__ void
test_initializers(TrackInitializerPointers initializers, double* output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id();
    if (thread_id.get() < *initializers.size)
    {
        TrackInitializer& init = initializers.storage[thread_id.get()];
        output += thread_id.get();
        *output = init.particle.energy;
    }
}

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
class TrackInitTest : public celeritas::Test
{
  protected:
    void SetUp() override
    {
        input.num_threads = 8;

        // Set up shared standard model particle data
        ParticleParams::VecAnnotatedDefs defs
            = {{{"gamma", pdg::gamma()},
                {0, 0, ParticleDef::stable_decay_constant()}}};
        particle_params = std::make_shared<ParticleParams>(std::move(defs));
        input.pparams   = particle_params->device_pointers();

        // Create primary particles
        for (unsigned long int i = 0; i < input.num_threads; ++i)
        {
            Primary p;
            p.def_id    = ParticleDefId{0};
            p.energy    = 10. * (i + 1);
            p.position  = {0., 0., 0.};
            p.direction = {0., 0., 1.};
            p.event_id  = EventId{0};
            host_primaries.push_back(p);
        }
        primaries = host_primaries;

        // Set the indices of the empty slots in the vector of track states; at
        // the start they are all empty
        for (auto i = 0; i < input.num_threads; ++i)
        {
            host_vacancies.push_back(i);
        }
        vacancies = host_vacancies;

        // Allocate memory for interactions
        interactions.resize(input.num_threads);
    }

    ITTestInput                        input;
    std::shared_ptr<ParticleParams>    particle_params;
    thrust::host_vector<Primary>       host_primaries;
    thrust::device_vector<Primary>     primaries;
    thrust::host_vector<ull_int>       host_vacancies;
    thrust::device_vector<ull_int>     vacancies;
    thrust::device_vector<ull_int>     num_vacancies;
    thrust::device_vector<size_type>   num_secondaries;
    thrust::device_vector<Interaction> interactions;
};

void print_tracks(ITTestInput input)
{
    // Allocate memory for results
    thrust::device_vector<double> output(input.num_threads);

    // Launch a kernel to check the properties of the initialized tracks
    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(input.num_threads);
    test_tracks<<<params.grid_size, params.block_size>>>(
        input, raw_pointer_cast(output.data()));
    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    thrust::host_vector<double> host_output = output;
    // const double expected[] = {80., 70., 60., 50., 40., 30., 20., 10.};
    // EXPECT_VEC_SOFT_EQ(expected, host_output);

    // Print the energies of the primary particles that were initialized
    std::cout << "\nTrack energies:" << std::endl;
    for (double x : host_output)
    {
        std::cout << x << std::endl;
    }
}

void print_initializers(TrackInitializerStore& ti_store)
{
    // Allocate memory for results
    size_type                     num_init = ti_store.get_size();
    thrust::device_vector<double> output(num_init);

    // Launch another kernel to check the properties of the initialized
    // tracks
    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(num_init);
    test_initializers<<<params.grid_size, params.block_size>>>(
        ti_store.device_pointers(), raw_pointer_cast(output.data()));
    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    thrust::host_vector<double> host_output = output;

    // Print the energies of the secondaries that will be used to initialize
    // new tracks
    std::cout << "\nInitializer energies:" << std::endl;
    for (auto x : host_output)
    {
        std::cout << x << std::endl;
    }
}

TEST_F(TrackInitTest, run)
{
    using thrust::raw_pointer_cast;

    // Allocate memory for particle track states
    ParticleStateStore pstore(input.num_threads);
    input.pstates = pstore.device_pointers();

    // Allocate memory for secondaries
    int                     storage_size = 1024;
    SecondaryAllocatorStore sa_store(storage_size);
    input.sa_view = sa_store.device_pointers();

    // Allocate memory for track initializers
    TrackInitializerStore ti_store(storage_size);

    // Create track initializers on device from primary particles
    span<const Primary> primary_ptrs
        = {raw_pointer_cast(primaries.data()), primaries.size()};
    primary_initializers(primary_ptrs, ti_store);

    // Initialize the primary tracks on device
    initialize_tracks(vacancies,
                      ti_store,
                      input.pparams,
                      input.pstates,
                      input.gparams,
                      input.gstates);

    // Print the energy of the initialize tracks
    print_tracks(input);

    // Launch kernel to process interactions
    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(input.num_threads);
    interact<<<params.grid_size, params.block_size>>>(
        input, raw_pointer_cast(interactions.data()));
    CELER_CUDA_CHECK_ERROR();

    // Launch kernel to find the indices of the empty slots in track vector
    span<const Interaction> interaction_ptrs
        = {raw_pointer_cast(interactions.data()), interactions.size()};
    find_vacancies(interaction_ptrs, num_vacancies, vacancies);

    // Launch kernel to count the number of secondaries produced in each
    // interaction
    count_secondaries(interaction_ptrs, num_secondaries);

    // Launch a kernel to create track initializers from
    // interactions/secondaries
    secondary_initializers(num_secondaries, interaction_ptrs, ti_store);

    // Print the energies of the track initializers created from secondaries
    print_initializers(ti_store);

    // Initialize secondaries on device
    initialize_tracks(vacancies,
                      ti_store,
                      input.pparams,
                      input.pstates,
                      input.gparams,
                      input.gstates);

    // Print the energies of the newly initialized tracks
    print_tracks(input);
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
