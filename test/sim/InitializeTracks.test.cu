#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file InitializeTracks.test.cu
//---------------------------------------------------------------------------//
#include "sim/InitializeTracks.hh"
#include "InitializeTracks.test.hh"

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include "base/KernelParamCalculator.cuda.hh"

namespace celeritas_test
{
using namespace celeritas;

//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void
initialize_vacancies_kernel(size_type num_tracks, span<size_type> vacancies)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id().get();
    if (thread_id < num_tracks)
    {
        vacancies.data()[thread_id] = thread_id;
    }
}

__global__ void interact_kernel(size_type                  num_tracks,
                                ParticleParamsPointers     pparams,
                                ParticleStatePointers      pstates,
                                SecondaryAllocatorPointers secondaries,
                                Interaction*               interactions)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id();
    if (thread_id.get() < num_tracks)
    {
        ParticleTrackView      particle(pparams, pstates, thread_id);
        SecondaryAllocatorView allocate_secondaries(secondaries);

        // Allow the particle to interact and create secondaries
        Interactor interact(particle, allocate_secondaries);
        interactions[thread_id.get()] = interact();
    }
}

__global__ void tracks_test_kernel(size_type              num_tracks,
                                   ParticleParamsPointers pparams,
                                   ParticleStatePointers  pstates,
                                   double*                output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id();
    if (thread_id.get() < num_tracks)
    {
        ParticleTrackView particle(pparams, pstates, thread_id);
        output[thread_id.get()] = particle.energy().value();
    }
}

__global__ void
initializers_test_kernel(span<TrackInitializer> initializers, double* output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id().get();
    if (thread_id < initializers.size())
    {
        TrackInitializer& init = initializers.data()[thread_id];
        output[thread_id]      = init.particle.energy.value();
    }
}

__global__ void
vacancies_test_kernel(span<size_type> vacancies, size_type* output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id().get();
    if (thread_id < vacancies.size())
    {
        output[thread_id] = vacancies.data()[thread_id];
    }
}

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//

void initialize_vacancies(size_type num_tracks, VacancyStore& vacancies)
{
    vacancies.resize(num_tracks);
    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(num_tracks);
    initialize_vacancies_kernel<<<params.grid_size, params.block_size>>>(
        num_tracks, vacancies.device_pointers());

    CELER_CUDA_CHECK_ERROR();
}

void interact(size_type                  num_tracks,
              ParticleParamsPointers     pparams,
              ParticleStatePointers      pstates,
              SecondaryAllocatorPointers secondaries,
              span<Interaction>          interactions)
{
    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(num_tracks);
    interact_kernel<<<params.grid_size, params.block_size>>>(
        num_tracks, pparams, pstates, secondaries, interactions.data());

    CELER_CUDA_CHECK_ERROR();
}

std::vector<double> tracks_test(size_type              num_tracks,
                                ParticleParamsPointers pparams,
                                ParticleStatePointers  pstates)
{
    // Allocate memory for results
    thrust::device_vector<double> output(num_tracks);

    // Launch a kernel to check the properties of the initialized tracks
    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(num_tracks);
    tracks_test_kernel<<<params.grid_size, params.block_size>>>(
        num_tracks, pparams, pstates, thrust::raw_pointer_cast(output.data()));

    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    std::vector<double> host_output(num_tracks);
    thrust::copy(output.begin(), output.end(), host_output.begin());

    return host_output;
}

std::vector<double> initializers_test(TrackInitializerStore& initializers)
{
    // Allocate memory for results
    thrust::device_vector<double> output(initializers.size());

    // Launch a kernel to check the properties of the track initializers
    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(initializers.size());
    initializers_test_kernel<<<params.grid_size, params.block_size>>>(
        initializers.device_pointers(),
        thrust::raw_pointer_cast(output.data()));

    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    std::vector<double> host_output(initializers.size());
    thrust::copy(output.begin(), output.end(), host_output.begin());

    return host_output;
}

std::vector<size_type> vacancies_test(VacancyStore& vacancies)
{
    // Allocate memory for results
    thrust::device_vector<size_type> output(vacancies.size());

    // Launch a kernel to check the indices of the empty slots
    KernelParamCalculator calc_launch_params;
    auto                  params = calc_launch_params(vacancies.size());
    vacancies_test_kernel<<<params.grid_size, params.block_size>>>(
        vacancies.device_pointers(), thrust::raw_pointer_cast(output.data()));

    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    std::vector<size_type> host_output(vacancies.size());
    thrust::copy(output.begin(), output.end(), host_output.begin());

    return host_output;
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
