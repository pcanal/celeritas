#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file InitializeTracks.test.cu
//---------------------------------------------------------------------------//
#include "sim/InitializeTracks.hh"
#include "InitializeTracks.test.hh"

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include "base/KernelParamCalculator.cuda.hh"

namespace celeritas_test
{
using namespace celeritas;

//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void interact_kernel(StatePointers              states,
                                ParamPointers              params,
                                SecondaryAllocatorPointers secondaries)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id();
    if (thread_id.get() < states.size())
    {
        ParticleTrackView particle(params.particle, states.particle, thread_id);
        SimTrackView      sim(states.sim, thread_id);
        SecondaryAllocatorView allocate_secondaries(secondaries);

        // Allow the particle to interact and create secondaries
        Interactor interact(particle, allocate_secondaries);
        states.interactions[thread_id.get()] = interact();

        // Mark the track as dead if the particle was killed
        if (action_killed(states.interactions[thread_id.get()].action))
        {
            sim.alive() = false;
        }
    }
}

__global__ void
tracks_test_kernel(StatePointers states, ParamPointers params, double* output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id();
    if (thread_id.get() < states.size())
    {
        ParticleTrackView particle(params.particle, states.particle, thread_id);
        output[thread_id.get()] = particle.energy().value();
    }
}

__global__ void
initializers_test_kernel(TrackInitializerPointers inits, double* output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id().get();
    if (thread_id < inits.initializers.size())
    {
        TrackInitializer& track = inits.initializers[thread_id];
        output[thread_id]       = track.particle.energy.value();
    }
}

__global__ void
vacancies_test_kernel(TrackInitializerPointers inits, size_type* output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id().get();
    if (thread_id < inits.vacancies.size())
    {
        output[thread_id] = inits.vacancies[thread_id];
    }
}

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//

void interact(StatePointers              states,
              ParamPointers              params,
              SecondaryAllocatorPointers secondaries)
{
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(states.size());
    interact_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, params, secondaries);

    CELER_CUDA_CHECK_ERROR();
}

std::vector<double> tracks_test(StatePointers states, ParamPointers params)
{
    // Allocate memory for results
    thrust::device_vector<double> output(states.size());

    // Launch a kernel to check the properties of the initialized tracks
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(states.size());
    tracks_test_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, params, thrust::raw_pointer_cast(output.data()));

    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    std::vector<double> host_output(states.size());
    thrust::copy(output.begin(), output.end(), host_output.begin());

    return host_output;
}

std::vector<double> initializers_test(TrackInitializerPointers inits)
{
    // Allocate memory for results
    thrust::device_vector<double> output(inits.initializers.size());

    // Launch a kernel to check the properties of the track initializers
    KernelParamCalculator calc_launch_params;
    auto lparams = calc_launch_params(inits.initializers.size());
    initializers_test_kernel<<<lparams.grid_size, lparams.block_size>>>(
        inits, thrust::raw_pointer_cast(output.data()));

    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    std::vector<double> host_output(inits.initializers.size());
    thrust::copy(output.begin(), output.end(), host_output.begin());

    return host_output;
}

std::vector<size_type> vacancies_test(TrackInitializerPointers inits)
{
    // Allocate memory for results
    thrust::device_vector<size_type> output(inits.vacancies.size());

    // Launch a kernel to check the indices of the empty slots
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(inits.vacancies.size());
    vacancies_test_kernel<<<lparams.grid_size, lparams.block_size>>>(
        inits, thrust::raw_pointer_cast(output.data()));

    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    std::vector<size_type> host_output(inits.vacancies.size());
    thrust::copy(output.begin(), output.end(), host_output.begin());

    return host_output;
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
