#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file InitializeTracks.test.cu
//---------------------------------------------------------------------------//
#include "sim/InitializeTracks.hh"
#include "InitializeTracks.test.hh"

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include "base/KernelParamCalculator.cuda.hh"

namespace celeritas_test
{
using namespace celeritas;

//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void interact_kernel(StatePointers              states,
                                ParamPointers              params,
                                SecondaryAllocatorPointers secondaries)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id();
    if (thread_id.get() < states.size())
    {
        ParticleTrackView particle(params.particle, states.particle, thread_id);
        SimTrackView      sim(states.sim, thread_id);
        SecondaryAllocatorView allocate_secondaries(secondaries);

        // Allow the particle to interact and create secondaries
        Interactor interact(particle, allocate_secondaries);
        states.interactions[thread_id.get()] = interact();

        // Mark the track as dead if the particle was killed
        if (action_killed(states.interactions[thread_id.get()].action))
        {
            sim.alive() = false;
        }
    }
}

__global__ void
tracks_test_kernel(StatePointers states, ParamPointers params, double* output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id();
    if (thread_id.get() < states.size())
    {
        ParticleTrackView particle(params.particle, states.particle, thread_id);
        output[thread_id.get()] = particle.energy().value();
    }
}

__global__ void
initializers_test_kernel(TrackInitializerPointers initializers, double* output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id().get();
    if (thread_id < initializers.tracks.size())
    {
        TrackInitializer& init = initializers.tracks[thread_id];
        output[thread_id]      = init.particle.energy.value();
    }
}

__global__ void
vacancies_test_kernel(TrackInitializerPointers initializers, size_type* output)
{
    auto thread_id = celeritas::KernelParamCalculator::thread_id().get();
    if (thread_id < initializers.vacancies.size())
    {
        output[thread_id] = initializers.vacancies[thread_id];
    }
}

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//

void interact(StatePointers              states,
              ParamPointers              params,
              SecondaryAllocatorPointers secondaries)
{
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(states.size());
    interact_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, params, secondaries);

    CELER_CUDA_CHECK_ERROR();
}

std::vector<double> tracks_test(StatePointers states, ParamPointers params)
{
    // Allocate memory for results
    thrust::device_vector<double> output(states.size());

    // Launch a kernel to check the properties of the initialized tracks
    KernelParamCalculator calc_launch_params;
    auto                  lparams = calc_launch_params(states.size());
    tracks_test_kernel<<<lparams.grid_size, lparams.block_size>>>(
        states, params, thrust::raw_pointer_cast(output.data()));

    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    std::vector<double> host_output(states.size());
    thrust::copy(output.begin(), output.end(), host_output.begin());

    return host_output;
}

std::vector<double> initializers_test(TrackInitializerPointers initializers)
{
    // Allocate memory for results
    thrust::device_vector<double> output(initializers.tracks.size());

    // Launch a kernel to check the properties of the track initializers
    KernelParamCalculator calc_launch_params;
    auto lparams = calc_launch_params(initializers.tracks.size());
    initializers_test_kernel<<<lparams.grid_size, lparams.block_size>>>(
        initializers, thrust::raw_pointer_cast(output.data()));

    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    std::vector<double> host_output(initializers.tracks.size());
    thrust::copy(output.begin(), output.end(), host_output.begin());

    return host_output;
}

std::vector<size_type> vacancies_test(TrackInitializerPointers initializers)
{
    // Allocate memory for results
    thrust::device_vector<size_type> output(initializers.vacancies.size());

    // Launch a kernel to check the indices of the empty slots
    KernelParamCalculator calc_launch_params;
    auto lparams = calc_launch_params(initializers.vacancies.size());
    vacancies_test_kernel<<<lparams.grid_size, lparams.block_size>>>(
        initializers, thrust::raw_pointer_cast(output.data()));

    CELER_CUDA_CHECK_ERROR();

    // Copy data back to host
    std::vector<size_type> host_output(initializers.vacancies.size());
    thrust::copy(output.begin(), output.end(), host_output.begin());

    return host_output;
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
